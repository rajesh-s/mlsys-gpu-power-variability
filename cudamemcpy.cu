#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <cmath>
#include "gputimer.h"
#include <string>
#include <sys/stat.h>
#include <fstream>
#include <hip/hip_runtime_api.h>
using namespace std;

// Function to read data from file
float* read_from_file(string file_name){
  // lets get filesize
  struct stat results;
  if (stat(file_name.c_str(), &results) != 0){
    // An error occurred
    std::cout << "ERROR: unable to get filesize" << std::endl;
    return NULL;
  }
  // The size of the file in bytes is in results.st_size
  // Lets allocate an array to contain the binary file
  //std::cout << "Filename: " << file_name << "Size:" <<  results.st_size << std::endl;
  float* data = (float *)malloc(results.st_size);


  // lets write it to binary file
  ifstream infile;

  // open a binary file
  infile.open(file_name, ios::binary | ios::in);

  //read data from file
  infile.read((char*) data, results.st_size);

  // close the file
  infile.close();

  return data;
}



// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Randomization helpers
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void fill_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}


void fill_cos(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(float(i + j * nr_rows_A));
}

int def(int value, int device) {

  hipSetDevice(device);
  hipStream_t computeStream;
  hipError_t result;
  result = hipStreamCreate(&computeStream);

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = value;

        float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	float *d_A;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));

  string fname_A = string("host_A_") + to_string(value) +string(".bin");
  float *h_A = read_from_file(fname_A);

  GpuTimer timer;
    timer.Start();
	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpyAsync(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, computeStream);
	//std::cout << "A =" << std::endl;
	//print_matrix(h_A, nr_rows_A, nr_cols_A);
    timer.Stop();
	std::cout <<"CudaMemCpy " << i << " Runtime = " << timer.Elapsed() << std::endl;
  }
  hipProfilerStop();

	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	//std::cout << "C =" << std::endl;
	//print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	hipFree(d_A);

  result = hipStreamDestroy(computeStream);

	// Free CPU memory
	free(h_A);
	free(h_C);

	return 0;
}

int main(int argc, char* argv[]){
	// for (int i=100; i <= 100000; i = i*10){
	// 	std::cout << "\n\n\n" << i << "\n";
	// 	def(1024, i);
	// }
	if (argc != 4){
		std::cout << "Usage: mul <dim> <target-device num>" << std::endl;
		exit(-1);
	}
	int dim = atoi(argv[1]);
	int device = atoi(argv[2]);
	//cout << dim <<
	def(dim, device);
	return 0;
}
