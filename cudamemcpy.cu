#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <cmath>
#include "gputimer.h"
#include <string>
#include <sys/stat.h>
#include <fstream>
using namespace std;

// Function to read data from file
float* read_from_file(string file_name){
  // lets get filesize
  struct stat results;
  if (stat(file_name.c_str(), &results) != 0){
    // An error occurred
    std::cout << "ERROR: unable to get filesize" << std::endl;
    return NULL;
  }
  // The size of the file in bytes is in results.st_size
  // Lets allocate an array to contain the binary file
  //std::cout << "Filename: " << file_name << "Size:" <<  results.st_size << std::endl;
  float* data = (float *)malloc(results.st_size);


  // lets write it to binary file
  ifstream infile;

  // open a binary file
  infile.open(file_name, ios::binary | ios::in);

  //read data from file
  infile.read((char*) data, results.st_size);

  // close the file
  infile.close();

  return data;
}



// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Randomization helpers
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void fill_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}


void fill_cos(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(float(i + j * nr_rows_A));
}

int def(int value, int reps, int device) {

  hipSetDevice(device);
  hipStream_t computeStream;
  hipError_t result;
  result = hipStreamCreate(&computeStream);

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = value;

        float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	float *d_A, *d_B;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B,nr_rows_A * nr_cols_A * sizeof(float));

  string fname_A = string("host_A_") + to_string(value) +string(".bin");
  float *h_A = read_from_file(fname_A);
	for(int i=1; i<=reps; i++){
  GpuTimer timer;
    timer.Start();
	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpyAsync(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, computeStream);
	hipMemcpyAsync(d_B,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, computeStream);
	//std::cout << "A =" << std::endl;
	//print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout <<"CudaMemCpy Runtime = " << timer.Elapsed() << std::endl;
	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	hipMemcpyAsync(h_C,d_B,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	//std::cout << "C =" << std::endl;
	//print_matrix(h_C, nr_rows_C, nr_cols_C);
    timer.Stop();
	}
	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);

  result = hipStreamDestroy(computeStream);

	// Free CPU memory
	free(h_A);
	free(h_C);

	return 0;
}

int main(int argc, char* argv[]){
	// for (int i=100; i <= 100000; i = i*10){
	// 	std::cout << "\n\n\n" << i << "\n";
	// 	def(1024, i);
	// }
	if (argc != 4){
		std::cout << "Usage: mul <dim> <target-device num>" << std::endl;
		exit(-1);
	}
	int dim = atoi(argv[1]);
	int reps = atoi(argv[2]);
	int device = atoi(argv[3]);
	def(dim, reps, device);
	return 0;
}
