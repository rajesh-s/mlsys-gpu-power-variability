// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cmath>


// Randomization helpers 
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void fill_sin(double *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}


void fill_cos(double *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(float(i + j * nr_rows_A));
}

#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul( hipblasHandle_t handle, const double *A, const double *B, double *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	// Do the actual multiplication
  hipblasStatus_t err = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
  if (err != HIPBLAS_STATUS_SUCCESS)
  	std::cout << "Error: " <<  _cudaGetErrorEnum(err) << std::endl;
  
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const double *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int def(int value, int reps, int device) {

  hipSetDevice(device);
  hipStream_t computeStream;
  hipError_t result;
  result = hipStreamCreate(&computeStream);

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = value;
	
	double *h_A = (double *)malloc(nr_rows_A * nr_cols_A * sizeof(double));
	double *h_B = (double *)malloc(nr_rows_B * nr_cols_B * sizeof(double));
	double *h_C = (double *)malloc(nr_rows_C * nr_cols_C * sizeof(double));

	// Allocate 3 arrays on GPU
	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(double));
	hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(double));
	hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(double));

	// cudaMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),cudaMemcpyHostToDevice);

	// Fill the arrays A and B on GPU with random numbers
	fill_sin(h_A, nr_rows_A, nr_cols_A);
	fill_cos(h_B, nr_rows_B, nr_cols_B);

	// We have useful values in A and B you can copy them in GPU:
	hipMemcpyAsync(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(double),hipMemcpyHostToDevice, computeStream);
	hipMemcpyAsync(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(double),hipMemcpyHostToDevice, computeStream);
	std::cout << "A =" << std::endl;
	std::cout << "B =" << std::endl;

  // Create a handle for CUBLAS
	hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetStream(handle, computeStream);

  for (int i=0; i< reps; i++){
	// Multiply A and B on GPU
  gpu_blas_mmul(handle, d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
  
  hipStreamSynchronize(computeStream);  
  }

	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(double),hipMemcpyDeviceToHost, computeStream);
	std::cout << "C =" << std::endl;
	// print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
    hipFree(d_C);
  
  result = hipStreamDestroy(computeStream);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}

int main(int argc, char* argv[]){
	// for (int i=100; i <= 100000; i = i*10){
	// 	std::cout << "\n\n\n" << i << "\n"; 
	// 	def(1024, i);
	// }
	if (argc != 4){
		std::cout << "Usage: mul <dim> <reps> <target-device num>" << std::endl;
		exit(-1);
	}
	int dim = atoi(argv[1]);
	int reps = atoi(argv[2]);
	int device = atoi(argv[3]);
	//cout << dim <<
	def(dim, reps, device);
	return 0;
}
